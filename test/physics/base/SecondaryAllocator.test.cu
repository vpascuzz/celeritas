#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file SecondaryAllocator.test.cu
//---------------------------------------------------------------------------//
#include "SecondaryAllocator.test.hh"

#include <thrust/device_vector.h>
#include "base/KernelParamCalculator.cuda.hh"

using thrust::raw_pointer_cast;

namespace celeritas_test
{
//---------------------------------------------------------------------------//
// KERNELS
//---------------------------------------------------------------------------//

__global__ void sa_test_kernel(SATestInput input)
{
    unsigned int local_thread_id
        = celeritas::KernelParamCalculator::thread_id().get();
    if (local_thread_id >= input.num_threads)
        return;

    num_allocations[local_thread_id] = 0;

    StackAllocator allocate(input.sa_view);
    for (int i = 0; i < input.num_iters; ++i)
    {
        void* new_data = allocate(input.alloc_size);
        if (new_data)
        {
            ++num_allocations[local_thread_id];
        }
    }
}

//---------------------------------------------------------------------------//
// TESTING INTERFACE
//---------------------------------------------------------------------------//
//! Run on device and return results
SATestOutput sa_test(SATestInput input)
{
    celeritas::KernelParamCalculator calc_launch_params;
    auto params = calc_launch_params(input.num_threads);
    sa_test_kernel<<<params.grid_size, params.block_size>>>(input.num_threads);

    SATestOutput result;
    return result;
}

//---------------------------------------------------------------------------//
} // namespace celeritas_test
